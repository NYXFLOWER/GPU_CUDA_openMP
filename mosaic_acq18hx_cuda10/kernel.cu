#include "hip/hip_runtime.h"
#include "mosaic_header.cuh"

#define FAILURE 0
#define SUCCESS !FAILURE

#define USER_NAME "acq18hx"

#define BUFFER_SIZE 100

// =========================== global variables ===========================
MODE execution_mode = CPU;
PPM write_type = PPM_BINARY;
char * in_file;
char * out_file;
char out_name[100];
int cell_size = 8;



// =========================== main ===========================
int 
main(int argc, char *argv[]) {
	//========================================================
	if (process_command_line(argc, argv) == FAILURE)
		return 1;
	//========================================================	

    // variable define and initialise
	Img * image;
	Mosaic * mos;
	unsigned long long int * ave;					// the average r, g, b over the whole image

    
	// allocate CPU memory
	image = (Img *)malloc(sizeof(Img));
	mos = (Mosaic *)malloc(sizeof(Mosaic));
	ave = (unsigned long long int *)malloc(3 * sizeof(unsigned long long int));

	
	// load image and get mosaic infomation
	read_ppm(in_file, image);
	compute_mosaic_info(cell_size, image, mos);


	// ------------- execute the mosaic filter based on the mode
	switch (execution_mode)
	{
	case CPU: {
	
		break;
	}

	case OPENMP: {
		break;
	}
		
	case GPU: {
		

		break;
	}

	case ALL: {
		/****** CPU ******/
		// start timing
		clock_t timer = clock();

		// process image
		run_cpu(image, mos, &process_mosaic_section_cpu);

		// compute and print time
		double cost = (double)(clock() - timer) / CLOCKS_PER_SEC;
		printf("CPU mode execution time took %d s and %.4fms\n\n", (int)cost, ((cost - (int)cost) * 1000));


		/****** OPENMP ******/
		// load image and get mosaic infomation
		read_ppm(in_file, image);
		compute_mosaic_info(cell_size, image, mos);

		// start timing
		double time_begin = omp_get_wtime();
		
		// process image
		run_cpu(image, mos, &process_mosaic_section_openmp2);

		// compute and print time
		double time_end = omp_get_wtime();
		cost = (double)(time_end - time_begin);
		printf("OPENMP mode execution time took %d s and %.4fms\n\n", (int)cost, (cost - (int)cost) * 1000);

		/****** GPU ******/
		// timing in the gpu image processing function

		// load image and get mosaic infomation
		read_ppm(in_file, image);
		compute_mosaic_info(cell_size, image, mos);

		// process image
		run_gpu(image, mos, ave);

		break;
	}

	default:
		break;
	}

	// ------------ write output image file (either binary or plain text ppm)
	sprintf(out_name, "%s_acq18hx.ppm", out_file);
	switch (write_type)
	{
	case PPM_PLAIN_TEXT: {
		write_ppm_text(image, out_name);
		break;
	}
	default: {
		write_ppm_binary(image, out_name);
		break;
	}
	}


	
	// free CPU memory
	free(ave);
	free(image->data);
	free(image);
	free(mos);

    return 0;
}

/************************ Various Utility Functions ****************************/
void 
print_help() {
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		"\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP or\n"
		"\t               ALL. The mode specifies which version of the simulation\n"
		"\t               code should execute. ALL should execute each mode in\n"
		"\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		"\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		"\t               PPM_PLAIN_TEXT\n ");
}

/* read and check the input parameters */
int 
process_command_line(int argc, char *argv[]) {
	if (argc < 7) {
		fprintf(stderr, "Error: Missing program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}
	//first argument is always the executable name

	//read in the non optional command line arguments
	cell_size = (unsigned int)atoi(argv[1]);
	if (!is_exp_of_two(cell_size)) {
		fprintf(stderr, "mosaic size should be exp of 2");
		print_help();
	}


	if (!strcmp(argv[2], "CPU")) { execution_mode = CPU; };
	if (!strcmp(argv[2], "OPENMP")) { execution_mode = OPENMP; };
	if (!strcmp(argv[2], "GPU")) { execution_mode = GPU; };
	if (!strcmp(argv[2], "ALL")) { execution_mode = ALL; };
	//TODO: read in the input image name
	in_file = argv[4];
	//TODO: read in the output image name
	out_file = argv[6];
	if (argc > 7) {
		//TODO: read in any optional part 3 arguments
		if (!strcmp(argv[8], "PPM_BINARY")) { write_type = PPM_BINARY; };
		if (!strcmp(argv[8], "PPM_PLAIN_TEXT")) { write_type = PPM_PLAIN_TEXT; };
	}

	return SUCCESS;
}

/* print error message and exit */
void
error(char *message) {
	fprintf(stderr, "ppm: %s\n", message);
	exit(1);
}

/** check if 2 ** n */
int
is_exp_of_two(unsigned int x) {
	unsigned int t = 1;
	while (1)
	{
		if (t == x) return 1;

		if (t > x) return 0;

		t = t * 2;
	}
}

/************************ PPM I/O Functions ****************************/
static void
skip_comment(FILE *fp) {
	int c = getc(fp);
	while (c == '#') {
		while (getc(fp) != '\n');
		c = getc(fp);
	}
	ungetc(c, fp);
}

/* read either p3 or p6, the output pixel data is a list of unsigned char */
void
read_ppm(const char *img_path, Img *image) {
	// open file
	FILE *fp = fopen(img_path, "rb");
	if (!fp) fprintf(stderr, "Image file cannot be accessed");

	// local variables
	unsigned int length;
	char buffer[BUFFER_SIZE];

	// get ppm type
	if (!fgets(buffer, BUFFER_SIZE, fp)) fprintf(stderr, "Cant get ppm type");
	PPM ppm_type = (buffer[1] == '6') ? PPM_BINARY : PPM_PLAIN_TEXT;

	// get comment and other three features
	skip_comment(fp);
	if (!fscanf(fp, "%d\n", &image->width)) fprintf(stderr, "fail to scan width");
	skip_comment(fp);
	if (!fscanf(fp, "%d\n", &image->height)) fprintf(stderr, "fail to scan height");
	skip_comment(fp);
	if (!fscanf(fp, "%d\n", &image->color_value)) fprintf(stderr, "fail to scan color range");


	image->num_pixel = image->width * image->height;
	length = 3 * image->num_pixel;

	// get pixel data
	switch (ppm_type) {
	case (PPM_BINARY): {
		image->data = (unsigned char *)malloc(length);
		if (!image->data) fprintf(stderr, "data cannot be allocated on memory");

		if (fread((void *)image->data, 1, (size_t)length, fp) != length)
			fprintf(stderr, "cannot read image data from file");
		//            printf("\n");
		//
		//            for(int i = 0; i<100; i++)
		//                printf("%hhu ", image->data[i]);
		break;
	}
	case (PPM_PLAIN_TEXT): {
		int temp_int;
		unsigned char * temp_data = image->data = (unsigned char *)malloc(length);
		int data_index = 0;
		while (fscanf(fp, "%d", &temp_int) == 1) {
			temp_data[data_index++] = (unsigned char)temp_int;
			//printf("%hhu %d \n", temp_data[data_index - 1], i++);
		}
		break;
	}
	}

	fclose(fp);
}

void
write_ppm_binary(Img * image, char * file_name) {
	FILE *fp = fopen(file_name, "wb");
	if (!fp) fprintf(stderr, "file cannot be created or opened");

	// write header
	fprintf(fp, "P6\n%d\n%d\n%d\n", image->width, image->height, image->color_value);

	// write pixel data
	unsigned int length = 3 * image->width * image->height;
	if (fwrite((void *)image->data, 1, (size_t)length, fp) != length)
		fprintf(stderr, "data cannot be written to file");

	// close file
	fclose(fp);
}

void
write_ppm_text(Img * image, char * file_name) {
	FILE *fp = fopen(file_name, "w");

	//  write header
	fprintf(fp, "P3\n%d %d\n%d\n", image->width, image->height, image->color_value);

	// format all data into string and write it into file 
	unsigned int index = 0;
	for (unsigned int i = 0; i < image->height; i++) {
		index = i * image->width * 3;
		for (unsigned int j = 0; j < (image->width - 1) * 3; j += 3) {
			fprintf(fp, "%d %d %d\t", image->data[index + j], image->data[index + j + 1], image->data[index + j + 2]);
		}

		index += (image->width - 1) * 3;
		if (i == image->height - 1) {
			fprintf(fp, "%d %d %d", image->data[index], image->data[index + 1], image->data[index + 2]);
		}
		else {
			fprintf(fp, "%d %d %d\n", image->data[index], image->data[index + 1], image->data[index + 2]);
		}

	}
	/**/

	fclose(fp);
}

/************************ Mosaic Function ****************************/
void
compute_mosaic_info(unsigned int cell_size, Img * image, Mosaic * mos) {
	mos->cell_size = cell_size;

	// check the cell size
	if (cell_size > image->height || cell_size > image->width)
	{
		printf("Warning: invalid mosaic size input!");
		exit(1);
	}

	// the numbers of mosaic cell in each row and column, as well as their remain pixels
	mos->cell_num_height = image->height / cell_size;
	mos->cell_remain_height = image->height % cell_size;
	mos->cell_num_weight = image->width / cell_size;
	mos->cell_remain_weight = image->width % cell_size;

	// calculate the total number of mosaic cell
	mos->cell_main_num = mos->cell_num_height * mos->cell_num_weight;
	mos->cell_num = mos->cell_main_num;
	if (mos->cell_remain_weight != 0) mos->cell_num += mos->cell_num_height;
	if (mos->cell_remain_height != 0) mos->cell_num += mos->cell_num_weight + 1;

	// calculate the number of pixel in each mosaic cell
	mos->pixcel_num = cell_size * cell_size;

}

/************************ GPU Mosaic Function ****************************/
__device__ unsigned long long int dev_ave[3];

 //========== kernel functions ============
__global__ void
averageFilterMain(unsigned int width, Mosaic * dev_mos, unsigned char * dev_data) {
	__shared__ unsigned char r, g, b;					// average over mosaic cell
	__shared__ unsigned int cell_r, cell_g, cell_b;		// sum over mosaic cell
	cell_r = 0; cell_g = 0; cell_b = 0;

	register int row_r = 0, row_g = 0, row_b = 0;		// sum over row in the cell


	register int index = ((blockIdx.x / dev_mos->cell_num_weight * dev_mos->cell_size + threadIdx.x) * width + (blockIdx.x % dev_mos->cell_num_weight) * dev_mos->cell_size) * 3;

	for (int i = 0; i < dev_mos->cell_size; i++) {
		row_r += (int)dev_data[index + i * 3];
		row_g += (int)dev_data[index + i * 3 + 1];
		row_b += (int)dev_data[index + i * 3 + 2];
	}

	// sum over the mosaic cell
	atomicAdd(&cell_r, row_r);
	atomicAdd(&cell_g, row_g);
	atomicAdd(&cell_b, row_b);

	//printf("from block %d, sum over image is %u, %u, %u \n", blockIdx.x, cell_r, cell_g, cell_b);
	//synchronize the local threads writing for average calculation
	__syncthreads();



	// calculate the average over the mosaic cell
	if (threadIdx.x == 0) {
		//printf("from block %d, thread %d, sum over row is %u, %u, %u \n", 
		//blockIdx.x, threadIdx.x, cell_r, cell_g, cell_b);

		unsigned int temp = dev_mos->cell_size * dev_mos->cell_size;
		r = (unsigned char)(cell_r / temp);
		g = (unsigned char)(cell_g / temp);
		b = (unsigned char)(cell_b / temp);

		// sum over the blocks
		atomicAdd(&dev_ave[0], (int)r);
		atomicAdd(&dev_ave[1], (int)g);
		atomicAdd(&dev_ave[2], (int)b);
	}

	//synchronize the local threads writing for average calculation
	__syncthreads();

	// assign averaged value back to image data
	for (int i = 0; i < dev_mos->cell_size; i++) {
		dev_data[index + i * 3] = r;
		dev_data[index + i * 3 + 1] = g;
		dev_data[index + i * 3 + 2] = b;
	}

	/*if (threadIdx.x == 1)
	{
	printf("from block %d, thread 1 is %u, %u, %u \n", blockIdx.x, dev_data[index], dev_data[index +1], dev_data[index+2]);
	}*/
}

__global__ void
averageFilterMain2(unsigned int width, Mosaic * dev_mos, uchar3 * dev_data) {
	__shared__ unsigned char r, g, b;					// average over mosaic cell
	__shared__ unsigned int cell_r, cell_g, cell_b;		// sum over mosaic cell
	cell_r = 0; cell_g = 0; cell_b = 0;

	register int row_r = 0, row_g = 0, row_b = 0;		// sum over row in the cell
	register uchar3 pixcel;

	register int index = (blockIdx.x / dev_mos->cell_num_weight * dev_mos->cell_size + threadIdx.x) * width + (blockIdx.x % dev_mos->cell_num_weight) * dev_mos->cell_size;

	for (int i = 0; i < dev_mos->cell_size; i++) {
		pixcel = dev_data[index + i];
		row_r += (int)pixcel.x;
		row_g += (int)pixcel.y;
		row_b += (int)pixcel.z;
	}

	// sum over the mosaic cell
	atomicAdd(&cell_r, row_r);
	atomicAdd(&cell_g, row_g);
	atomicAdd(&cell_b, row_b);

	//printf("from block %d, sum over image is %u, %u, %u \n", blockIdx.x, cell_r, cell_g, cell_b);
	//synchronize the local threads writing for average calculation
	__syncthreads();



	// calculate the average over the mosaic cell
	if (threadIdx.x == 0) {
		//printf("from block %d, thread %d, sum over row is %u, %u, %u \n", 
		//blockIdx.x, threadIdx.x, cell_r, cell_g, cell_b);

		unsigned int temp = dev_mos->cell_size * dev_mos->cell_size;
		r = (unsigned char)(cell_r / temp);
		g = (unsigned char)(cell_g / temp);
		b = (unsigned char)(cell_b / temp);

		// sum over the blocks
		atomicAdd(&dev_ave[0], (int)r);
		atomicAdd(&dev_ave[1], (int)g);
		atomicAdd(&dev_ave[2], (int)b);
	}

	//synchronize the local threads writing for average calculation
	__syncthreads();

	// assign averaged value back to image data
	for (int i = 0; i < dev_mos->cell_size; i++) {
		pixcel.x = r;
		pixcel.y = g;
		pixcel.z = b;
		dev_data[index + i] = pixcel;
	}

	/*if (threadIdx.x == 1)
	{
	printf("from block %d, thread 1 is %u, %u, %u \n", blockIdx.x, dev_data[index], dev_data[index +1], dev_data[index+2]);
	}*/
}

// ========== run functions ============
void
run_gpu(Img * image, Mosaic * mos, unsigned long long int * ave) {
	// process: average filter on GPU
	// change:  outer variable: image
	// return:  cudaStatus

	// variable declear and initialise
	hipError_t cudaStatus = hipSuccess;
	hipEvent_t start, stop;
	float ms;						// timing

	Img * dev_image;
	Mosaic * dev_mos;
	unsigned char * dev_data;		// array of r, g, b in device


	// create timer
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// starting timing here
	hipEventRecord(start, 0);


	// allocate GPU memory
	cudaStatus = hipMalloc((void **)&dev_image, sizeof(Img));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc image failed!");
		exit(1);
	}

	cudaStatus = hipMalloc((void **)&dev_mos, sizeof(Mosaic));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc mosaic info failed!");
		hipFree(dev_image);
		exit(1);
	}

	cudaStatus = hipMalloc((void **)&dev_data, (3 * image->num_pixel));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc data failed!");
		hipFree(dev_image);
		hipFree(dev_mos);
		exit(1);
	}


	// copy data from host to device
	cudaStatus = hipMemcpy(dev_image, image, sizeof(Img), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(dev_mos, mos, sizeof(Img), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(dev_data, image->data, 3 * image->num_pixel * sizeof(unsigned char), hipMemcpyHostToDevice);

	// ======================= process mosaic =======================
	// -> main section <-
	unsigned int blocksPerGrid = mos->cell_num_weight * mos->cell_num_height;
	unsigned int threadsPerBlock = mos->cell_size;
	averageFilterMain << <blocksPerGrid, threadsPerBlock >> > (image->width, dev_mos, dev_data);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();

	// copy image data from device to host
	cudaStatus = hipMemcpy(image->data, dev_data, 3 * image->num_pixel * sizeof(unsigned char), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpyFromSymbol(ave, HIP_SYMBOL(dev_ave), 3 * sizeof(unsigned long long int));


	// print the average r, g and b over image
	printf("Average image colour red = %llu, green = %llu, blue = %llu \n",
		(ave[0] / mos->cell_num), (ave[1] / mos->cell_num), (ave[2] / mos->cell_num));

	
	// end timing and print
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	printf("CUDA mode execution time took %d s and %f ms\n", (int)ms / 1000, ms);


	// free cuda memory
	hipFree(dev_image);
	hipFree(dev_mos);
	hipFree(dev_data);


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}

}


void
run_gpu2(Img * image, Mosaic * mos, unsigned long long int * ave) {
	// process: average filter on GPU
	// change:  outer variable: image
	// return:  cudaStatus

	// variable declear and initialise
	hipError_t cudaStatus = hipSuccess;
	hipEvent_t start, stop;
	float ms;						// timing

	Img * dev_image;
	Mosaic * dev_mos;
	uchar3 * data3;
	uchar3 * dev_data;		// array of pixcel with r, g, b dimensions in device

	// change the data from 1d to 3d
	data3 = (uchar3*)malloc(image->num_pixel * sizeof(uchar3));
	for (unsigned int i = 0; i < image->num_pixel; i++) {
		data3[i].x = image->data[i * 3];
		data3[i].y = image->data[i * 3 + 1];
		data3[i].z = image->data[i * 3 + 2];
	}


	// create timer
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// starting timing here
	hipEventRecord(start, 0);


	// allocate GPU memory
	cudaStatus = hipMalloc((void **)&dev_image, sizeof(Img));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc image failed!");
		exit(1);
	}

	cudaStatus = hipMalloc((void **)&dev_mos, sizeof(Mosaic));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc mosaic info failed!");
		hipFree(dev_image);
		exit(1);
	}

	cudaStatus = hipMalloc((void **)&dev_data, image->num_pixel * sizeof(uchar3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc data failed!");
		hipFree(dev_image);
		hipFree(dev_mos);
		exit(1);
	}


	// copy data from host to device
	cudaStatus = hipMemcpy(dev_image, image, sizeof(Img), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(dev_mos, mos, sizeof(Img), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(dev_data, data3, image->num_pixel * sizeof(uchar3), hipMemcpyHostToDevice);
	// ======================= process mosaic =======================
	// -> main section <-
	unsigned int blocksPerGrid = mos->cell_num_weight * mos->cell_num_height;
	unsigned int threadsPerBlock = mos->cell_size;
	averageFilterMain2 << <blocksPerGrid, threadsPerBlock >> > (image->width, dev_mos, dev_data);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();

	// copy image data from device to host
	cudaStatus = hipMemcpy(dev_data, data3, image->num_pixel * sizeof(uchar3), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpyFromSymbol(ave, HIP_SYMBOL(dev_ave), 3 * sizeof(unsigned long long int));


	// print the average r, g and b over image
	printf("Average image colour red = %llu, green = %llu, blue = %llu \n",
		(ave[0] / mos->cell_num), (ave[1] / mos->cell_num), (ave[2] / mos->cell_num));


	// end timing and print
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	printf("CUDA mode execution time took %d s and %f ms\n", (int)ms / 1000, ms);


	// chage the data type back to 1d
	for (unsigned int i = 0; i < image->num_pixel; i++) {
		image->data[i * 3] = data3[i].x;
		image->data[i * 3 + 1] = data3[i].y;
		image->data[i * 3 + 2] = data3[i].z;
	}


	// free cpu memory
	free(data3);

	// free cuda memory
	hipFree(dev_image);
	hipFree(dev_mos);
	hipFree(dev_data);


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}

}

/************************ Index Function ****************************/
int
index_main(int cell, int row, Img * image, Mosaic* mos) {
	return ((cell / mos->cell_num_weight * mos->cell_size + row) * image->width + (cell % mos->cell_num_weight) * mos->cell_size) * 3;
}

int
index_column_edge(int cell, int row, Img * image, Mosaic * mos) {
	return (cell * image->width * mos->cell_size + row * image->width + mos->cell_num_weight * mos->cell_size) * 3;
}

int
index_row_edge(int cell, int row, Img * image, Mosaic * mos) {
	return (image->width * (mos->cell_num_height * mos->cell_size + row) + cell * mos->cell_size) * 3;
}

int
index_end_case(int cell, int row, Img * image, Mosaic * mos) {
	return ((mos->cell_num_height * mos->cell_size + row) * image->width + mos->cell_num_weight * mos->cell_size) * 3;
}


/************************ CPU Mosaic Function ****************************/

void
process_mosaic_section_cpu(Img * image, Mosaic * mos, int limits[3], Func fff, int pixcel_num, int * total) {
	/** limits[0]: the number of mosaic in this mosaic section
	limits[1]: the number of pixcel rows in the current mosaic
	limits[2]: the number of pixcel columns in the current mosaic*/

	for (int i = 0; i < limits[0]; i++)	// the ith cell
	{
		/****** variables declaraction ******/
		unsigned char r, g, b;
		int r_sum = 0, g_sum = 0, b_sum = 0;
		int index;

		for (int j = 0; j < limits[1]; j++)	// the jth row of ith cell
		{
			// get the index in photo data that strar the jth row of the ith cell
			index = fff(i, j, image, mos);

			for (int k = 0; k < limits[2]; k++)	// the kth element of jth row
			{
				r_sum += (int)image->data[index + k * 3];
				g_sum += (int)image->data[index + k * 3 + 1];
				b_sum += (int)image->data[index + k * 3 + 2];
			}
		}

		// calculate the average
		r = (unsigned char)(r_sum / pixcel_num);
		g = (unsigned char)(g_sum / pixcel_num);
		b = (unsigned char)(b_sum / pixcel_num);

		// add to total
		total[0] += r; total[1] += g; total[2] += b;

		// mosaic the original
		for (int j = 0; j < limits[1]; ++j) {      // the jth row of ith cell
												   // index in data that start the l row in ith cell
			index = fff(i, j, image, mos);

			for (int k = 0; k < limits[2]; ++k) {       // the kth element of jth row
				image->data[index + k * 3] = r;
				image->data[index + k * 3 + 1] = g;
				image->data[index + k * 3 + 2] = b;
			}
		}
	}
}

void
run_cpu(Img * image, Mosaic * mos, Process ppp) {
	/****** variables declaraction ******/
	int * limits = (int *)malloc(3);
	int pixcel_num;
	int * total = (int *)malloc(3);
	total[0] = 0; total[1] = 0; total[2] = 0;


	/****** process main mosaic section ******/
	limits[0] = mos->cell_main_num;
	limits[1] = mos->cell_size;
	limits[2] = mos->cell_size;

	pixcel_num = mos->cell_size * mos->cell_size;
	ppp(image, mos, limits, &index_main, pixcel_num, total);

	/****** process column-edge mosaic section ******/
	if (mos->cell_remain_weight != 0)
	{
		limits[0] = mos->cell_num_height;
		limits[1] = mos->cell_size;
		limits[2] = mos->cell_remain_weight;

		pixcel_num = mos->cell_size * mos->cell_remain_weight;

		ppp(image, mos, limits, &index_column_edge, pixcel_num, total);
	}

	/****** process row-edge mosaic section ******/
	if (mos->cell_remain_weight != 0)
	{
		limits[0] = mos->cell_num_weight;
		limits[1] = mos->cell_remain_height;
		limits[2] = mos->cell_size;

		pixcel_num = mos->cell_size * mos->cell_remain_height;
		ppp(image, mos, limits, &index_row_edge, pixcel_num, total);

		/****** process end-corner mosaic section ******/
		limits[0] = 1;
		limits[1] = mos->cell_remain_height;
		limits[2] = mos->cell_remain_weight;

		pixcel_num = mos->cell_remain_height * mos->cell_remain_weight;

		ppp(image, mos, limits, &index_end_case, pixcel_num, total);
		//process_mosaic_section_cpu(image, mos, limits, &index_end_case, pixcel_num, total);		// for CPU version 2.0

	}

	// Output the average colour value for the image
	printf("Average image colour red = %d, green = %d, blue = %d \n",
		total[0] / mos->cell_num, total[1] / mos->cell_num, total[2] / mos->cell_num);
}

/************************ OpenMP Mosaic Function ****************************/
void
process_mosaic_section_openmp2(Img * image, Mosaic * mos, int limits[3], Func fff, int pixcel_num, int * total) {

	/** limits[0]: the number of mosaic in this mosaic section
	limits[1]: the number of pixcel rows in the current mosaic
	limits[2]: the number of pixcel columns in the current mosaic*/
	int i = 0;
	omp_set_num_threads(16);

	// construct array of int for r, g, b, each thread can only access an element dependent on its thread number.
	int rrr[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int ggg[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int bbb[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	//#pragma omp parallel for private(i) schedule(runtime)
	//#pragma omp parallel for private(i) schedule(static, 1)
#pragma omp parallel for private(i) schedule(static)
	//#pragma omp parallel for private(i) schedule(dynamic)
	for (i = 0; i < limits[0]; i++)	// the ith cell
	{
		/****** variables declaraction ******/
		unsigned char r, g, b;
		int r_sum = 0, g_sum = 0, b_sum = 0;
		int index;


		for (int j = 0; j < limits[1]; j++)	// the jth row of ith cell
		{
			// get the index in photo data that strar the jth row of the ith cell
			index = fff(i, j, image, mos);

			for (int k = 0; k < limits[2]; k++)	// the kth element of jth row
			{
				r_sum += (int)image->data[index + k * 3];
				g_sum += (int)image->data[index + k * 3 + 1];
				b_sum += (int)image->data[index + k * 3 + 2];
			}
		}

		// calculate the average
		r = (unsigned char)(r_sum / pixcel_num);
		g = (unsigned char)(g_sum / pixcel_num);
		b = (unsigned char)(b_sum / pixcel_num);

		// add to total
		rrr[omp_get_thread_num()] += r;
		ggg[omp_get_thread_num()] += g;
		bbb[omp_get_thread_num()] += b;

		// mosaic the original
		for (int j = 0; j < limits[1]; ++j) {      // the jth row of ith cell
												   // index in data that start the l row in ith cell
			index = index = fff(i, j, image, mos);

			for (int k = 0; k < limits[2]; ++k) {       // the kth element of jth row
				image->data[index + k * 3] = r;
				image->data[index + k * 3 + 1] = g;
				image->data[index + k * 3 + 2] = b;
			}
		}

	}

	for (int m = 0; m < 16; m++)
	{
		total[0] += rrr[m];
		total[1] += ggg[m];
		total[2] += bbb[m];
	}
}