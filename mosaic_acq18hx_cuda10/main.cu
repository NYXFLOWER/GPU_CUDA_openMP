#include "hip/hip_runtime.h"
//
// Created by Hao Xu on 2019-03-07.
//
#include "mosaic_header.cuh"

#define FAILURE 0
#define SUCCESS !FAILURE

#define USER_NAME "acq18hx"        //replace with your user name

//========================================================
void print_help();
int process_command_line(int argc, char *argv[]);
//========================================================

MODE execution_mode = CPU;
PPM write_type = PPM_BINARY;
char * in_file;
char * out_file;
char out_name[100];
int cell_size = 8;

int main(int argc, char *argv[]) {
	//========================================================
	if (process_command_line(argc, argv) == FAILURE)
		return 1;
	//========================================================

	
	

	
	


	// ------------ read input image file (either binary or plain text ppm)
	// allocate memory
	Img * image = (Img *)malloc(sizeof(Img));
	if (!image) error("New image cannot be allocated on memory");
	Mosaic * mos = (Mosaic *)malloc(sizeof(Mosaic));

	// load image and get mosaic infomation
	read_ppm(in_file, image);
	compute_mosaic_info(cell_size, image, mos);

	// ------------- execute the mosaic filter based on the mode
	switch (execution_mode)
	{
	case CPU:
		break;
	case OPENMP:
		break;
	case GPU:
		//  in device
		Img * dev_image;
		Mosaic * dev_mos;

		// allocate GPU memory
		hipMalloc((void **)&dev_image, sizeof(Img));
		break;
	case ALL:
		break;
	default:
		break;
	}
	

	// free image and data
	

	return 0;
}

//========================================================
void print_help() {
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		"\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP or\n"
		"\t               ALL. The mode specifies which version of the simulation\n"
		"\t               code should execute. ALL should execute each mode in\n"
		"\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		"\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		"\t               PPM_PLAIN_TEXT\n ");
}


int process_command_line(int argc, char *argv[]) {
	if (argc < 7) {
		fprintf(stderr, "Error: Missing program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}
	//first argument is always the executable name

	//read in the non optional command line arguments
	cell_size = (unsigned int)atoi(argv[1]);
	if (!is_exp_of_two(cell_size)) {
		error("mosaic size should be exp of 2");
		print_help();
	}


	if (!strcmp(argv[2], "CPU")) { execution_mode = CPU; };
	if (!strcmp(argv[2], "OPENMP")) { execution_mode = OPENMP; };
	if (!strcmp(argv[2], "GPU")) { execution_mode = GPU; };
	if (!strcmp(argv[2], "ALL")) { execution_mode = ALL; };
	//TODO: read in the input image name
	in_file = argv[4];
	//TODO: read in the output image name
	out_file = argv[6];
	if (argc > 7) {
		//TODO: read in any optional part 3 arguments
		if (!strcmp(argv[8], "PPM_BINARY")) { write_type = PPM_BINARY; };
		if (!strcmp(argv[8], "PPM_PLAIN_TEXT")) { write_type = PPM_PLAIN_TEXT; };
	}
	
	return SUCCESS;
}

