//
// Created by Hao Xu on 2019-03-07.
//
#include "mosaic_header.cuh"

void
process_mosaic_section_cpu(Img * image, Mosaic * mos, int limits[3], Func fff, int pixcel_num, int * total) {
	/** limits[0]: the number of mosaic in this mosaic section
	limits[1]: the number of pixcel rows in the current mosaic
	limits[2]: the number of pixcel columns in the current mosaic*/

	for (int i = 0; i < limits[0]; i++)	// the ith cell
	{
		/****** variables declaraction ******/
		unsigned char r, g, b;
		int r_sum = 0, g_sum = 0, b_sum = 0;
		int index;

		for (int j = 0; j < limits[1]; j++)	// the jth row of ith cell
		{
			// get the index in photo data that strar the jth row of the ith cell
			index = fff(i, j, image, mos);

			for (int k = 0; k < limits[2]; k++)	// the kth element of jth row
			{
				r_sum += (int)image->data[index + k * 3];
				g_sum += (int)image->data[index + k * 3 + 1];
				b_sum += (int)image->data[index + k * 3 + 2];
			}
		}

		// calculate the average
		r = (unsigned char)(r_sum / pixcel_num);
		g = (unsigned char)(g_sum / pixcel_num);
		b = (unsigned char)(b_sum / pixcel_num);

		// add to total
		total[0] += r; total[1] += g; total[2] += b;

		// mosaic the original
		for (int j = 0; j < limits[1]; ++j) {      // the jth row of ith cell
												   // index in data that start the l row in ith cell
			index = fff(i, j, image, mos);

			for (int k = 0; k < limits[2]; ++k) {       // the kth element of jth row
				image->data[index + k * 3] = r;
				image->data[index + k * 3 + 1] = g;
				image->data[index + k * 3 + 2] = b;
			}
		}
	}
}

void
process_mosaic_s